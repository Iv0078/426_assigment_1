#include "hip/hip_runtime.h"
#include "utility_functions.cuh"


bool CheckCollision(Ball* one, Ball* two);
void resolveCollisions( int i, Ball* balls,  int* N);


void resolveCollision(Ball* a, int& weight);
void Move(Ball* a,float* dt,  int* window_width,  int* window_height);


__global__ void update_program_kernel(Ball* a,  int* N,  float* dt,  int* height,  int* width) {
    
    int i = threadIdx.x;
    if (i < *N) {
        printf("Integer: %d\n", i);
        Move(&(a[i]),dt, width, height);
        //resolveCollisions(i, a, N);
    }
    
    
        //printf("start\n" );
  
        
    
    
}

namespace Wrapper {
	void wrapper(Ball* balls, const int& N, const float& dt, const int& height, const  int& width )
	{

       /* for (int i{ 0 }; i < N; i++) {
            float x = (balls[i].position.x);
            printf("weidght %f\n",x );
        }*/

        //printf("start");
        /*for (int i{ 0 }; i < N; i++) {
            printf("weidght %i\n", balls[i].weight);
            
        }*/


        size_t sz = N * sizeof(Ball);
        

        int o_N = N;
        float o_dt = dt;
        int o_height = height;
        int o_width = width;



        
        //Ball** ball_inst;
        Ball* arr_d_a = new Ball[N];

        int* d_N;
        float* d_dt ;
        int* d_height;
        int* d_width;
        
        Ball* arr_f_a = new Ball[N];

        //hipMalloc(&ball_inst, 2*sz);
        hipMalloc((void**)&arr_d_a, sz);

        hipMalloc((void**)&d_N, sizeof(int));
        hipMalloc((void**)&d_dt, sizeof(float));
        hipMalloc((void**)&d_height, sizeof(int));
        hipMalloc((void**)&d_width, sizeof(int));


       
        hipMemcpy(arr_d_a,&balls, sz, hipMemcpyHostToDevice);
        hipMemcpy(d_N, &o_N, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_dt, &o_dt, sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_height, &o_height, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_width, &o_width, sizeof(int), hipMemcpyHostToDevice);


        update_program_kernel <<<1, N >> > (arr_d_a, d_N, d_dt, d_height, d_width);
        hipDeviceSynchronize();

        hipMemcpy(&balls, arr_d_a, sz, hipMemcpyDeviceToHost);

        


		hipFree(arr_d_a);

        hipFree(d_N);
        hipFree(d_dt);
        hipFree(d_height);
        hipFree(d_width);

   
        
	}
}



__host__ __device__  bool CheckCollision(Ball* one, Ball* two)
{
    const float adjt{ 4.0f };
    // collision x-axis?
    bool collisionX = (*one).position.x + (*one).size.x - adjt >= (*two).position.x &&
        (*two).position.x + (*two).size.x - adjt >= (*one).position.x;
    // collision y-axis?
    bool collisionY = (*one).position.y + (*one).size.y - adjt >= (*two).position.y &&
        (*two).position.y + (*two).size.y - adjt >= (*one).position.y;
    // collision only if on both axes
    return collisionX && collisionY;
}

__host__ __device__  void resolveCollisions( int i, Ball* balls,  int* N) {

    bool collision = false;
    int ballweight{ 5 };

    for (size_t j{ 0 }; j < *N; j++) {
        if (i != j) {
            collision = CheckCollision(&(balls[i]), (&balls[j]) );
        }
        if (collision) {
            ballweight = (balls[j]).weight;
            (balls[i]).collision = true;
            break;
        }
    }

    if ((balls[i]).getCollision()) {
        (balls[i]).resolveCollision(ballweight);
    }

}


__device__  void  resolveCollision(Ball * a,  int* weight) {
    if ((*a).collision) {
        float ratio = 1 - 2 * ((*a).weight - *weight) / 100;
        (*a).velocity.x = -ratio * (*a).velocity.x;
        (*a).velocity.y = -ratio * (*a).velocity.y;
        (*a).collision = false;
    }
    return;
}


__device__  void Move(Ball* a, float* dt,  int* window_width,  int* window_height) {
    /*std::cout << "x: " << position.x << std::endl;
    std::cout << "y: " << position.y << std::endl;
    std::cout << "x_vel: " << velocity.x << std::endl;
    std::cout << "yLvec: " << velocity.y << std::endl;
    velocity.x += 1;*/
  
        // move the ball
    //printf("vel: %f\n", (*a).velocity.x);
    (*a).position += (*a).velocity * (*dt);
    if ((*a).position.x <= 0.0f)
    {
        (*a).velocity.x = -(*a).velocity.x;
        (*a).position.x = 0.0f;
    }
    else if ((*a).position.x + (*a).size.x >= *window_width)
    {
        (*a).velocity.x = -(*a).velocity.x;
        (*a).position.x = *window_width - (*a).size.x;
    }
    if ((*a).position.y <= 0.0f)
    {
        (*a).velocity.y = -(*a).velocity.y;
        (*a).position.y = 0.0f;
    }
    else if ((*a).position.y + (*a).size.y >= *window_height)
    {
        (*a).velocity.y = -(*a).velocity.y;
        (*a).position.y = *window_height - (*a).size.y;
        /* (*a).velocity.y = 0;
            (*a).velocity.x = 0;
            (*a).stopped = true;*/
    }

    // gravity
    if (!((*a).position.y == *window_height && (*a).velocity.y == 0)) {
        (*a).velocity.y = (*a).velocity.y + (*a).gravity; // set to .25 for convinence
    }
    //printf("vel: %f\n", (*a).velocity.x);
    return;
}